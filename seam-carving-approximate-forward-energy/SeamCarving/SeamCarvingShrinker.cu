#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <iostream>
#include <cmath>

#include "SeamCarvingShrinker.h"

// TODO: Get based on GPU
#ifndef BLOCK_SIZE
# define BLOCK_SIZE 1024
#endif

#ifndef NUM_COLORS
# define NUM_COLORS 3
#endif

#ifndef IMG_VAL
# define IMG_VAL(arr, width, col, row, color) ((arr)[(NUM_COLORS * (width) * (row)) + ((col) * NUM_COLORS) + (color)])
#endif

#ifndef ENERGY_IMG_VAL
# define ENERGY_IMG_VAL(arr, width, col, row) ((arr)[((width) * (row)) + (col)])
#endif

using namespace cimg_library;
using namespace std;

inline float getPixelColorsEuclideanDifference(float * workingArray, int originalWidth,
  int i1, int j1, int i2, int j2)
{
    float sumSquare = 0.0;

    for (int color = 0; color < NUM_COLORS; color++) {
        float diff =
          abs(IMG_VAL(workingArray, originalWidth, i1, j1, color)
            - IMG_VAL(workingArray, originalWidth, i2, j2, color));
        sumSquare += diff;
    }

    return sqrt(sumSquare);
}

float getPixelEnergy(float * workingArray, int i, int j, int originalWidth, int originalHeight, int currentWidth,
  int currentHeight)
{
    float totalEnergy = 0.0;

    if (i > 0) {
        totalEnergy += getPixelColorsEuclideanDifference(workingArray, originalWidth, i, j, i - 1, j);
    }
    if (i < originalWidth - 1) {
        totalEnergy += getPixelColorsEuclideanDifference(workingArray, originalWidth, i, j, i + 1, j);
    }
    if (j > 0) {
        totalEnergy += getPixelColorsEuclideanDifference(workingArray, originalWidth, i, j, i, j - 1);
    }
    if (j < originalHeight - 1) {
        totalEnergy += getPixelColorsEuclideanDifference(workingArray, originalWidth,
            i, j, i, j + 1);
    }

    // forward energy stuff
    // we're going to calculate forward energy in both directions because figuring out which
    // direction to calculate forward energy for would require a large rewrite, and this
    // is an easy improvement
    if (i > 0 && i < originalWidth - 1) {
        totalEnergy += getPixelColorsEuclideanDifference(workingArray, originalWidth, i - 1, j, i + 1, j);
    }
    if (j > 0 && j < originalHeight - 1) {
        totalEnergy += getPixelColorsEuclideanDifference(workingArray, originalWidth, i, j - 1, i, j + 1);
    }

    return totalEnergy;
}

float * getWorkingArray(CImg<float> image)
{
    auto width  = image.width();
    auto height = image.height();

    // assume image depth of 1

    float * result;

    hipMallocManaged(&result, width * height * NUM_COLORS * sizeof(float)); // RGB image

    // old fashioned copy - not using Cuda since this isn't a bottleneck
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            for (int color = 0; color < NUM_COLORS; color++) {
                IMG_VAL(result, width, i, j, color) = image(i, j, 0, color);
            }
        }
    }

    return result;
}

float * getEnergyMap(float * workingArray, int originalWidth, int originalHeight, int currentWidth,
  int currentHeight)
{
    float * result;

    hipMallocManaged(&result, originalWidth * originalHeight * sizeof(float)); // RGB image

    // old fashioned copy - not using Cuda since this isn't a bottleneck
    for (int i = 0; i < originalWidth; i++) {
        for (int j = 0; j < originalHeight; j++) {
            ENERGY_IMG_VAL(result, originalWidth, i, j) = getPixelEnergy(
                workingArray, i, j, originalWidth, originalHeight, currentWidth, currentHeight);
        }
    }

    return result;
}

CImg<float> getOutputImage(float * workingArray, int originalWidth, int originalHeight, int newWidth, int newHeight) // also cleans up input array
{
    CImg<float> result(newWidth, newHeight, 1, NUM_COLORS);

    for (int i = 0; i < newWidth; i++) {
        for (int j = 0; j < newHeight; j++) {
            for (int color = 0; color < NUM_COLORS; color++) {
                result(i, j, 0, color) = IMG_VAL(workingArray, originalWidth, i, j, color);
            }
        }
    }

    return result;
}

CImg<float> getOutputEnergyMapImage(float * workingenergyMapArray, int originalWidth, int newWidth, int newHeight) // also cleans up input array
{
    CImg<float> result(newWidth, newHeight, 1, NUM_COLORS);

    for (int i = 0; i < newWidth; i++) {
        for (int j = 0; j < newHeight; j++) {
            result(i, j, 0, 0) = ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, i, j);
        }
    }

    return result;
}

__global__
void calculateVerticalCumulativeEnergyTableKernel(
    float * workingenergyMapArray,
    bool * hasFinishedCalculationTable,
    float * cumulativeEnergyTable,
    int originalWidth,
    int currentWidth, int currentHeight
)
{
    int index  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;


    for (int row = 0; row < currentHeight; row++) {
        // iterate over every column this kernel is responsible for
        for (int col = index; col < currentWidth; col += stride) {
            // calculate the minimum cumulative energy for this index;
            float energy = ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, col, row);
            if (row > 0) {
                float minPreviousEnergy  = ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, col, row - 1); // guaranteed to be done
                int numSideColumns       = 0;
                int sideColumnIndices[2] = { 0, 0 }; // placeholders
                if (col > 0) {
                    sideColumnIndices[numSideColumns] = col - 1;
                    numSideColumns++;
                }
                if (col < currentWidth - 1) {
                    sideColumnIndices[numSideColumns] = col + 1;
                    numSideColumns++;
                }
                bool dependenciesResolved = false;
                // keep checking to see if dependencies (adjacent columns) have been resolved
                while (!dependenciesResolved) {
                    bool newDependenciesResolved = true;
                    for (int i = 0; i < numSideColumns; i++) {
                        newDependenciesResolved = newDependenciesResolved &&
                          ENERGY_IMG_VAL(hasFinishedCalculationTable, originalWidth, sideColumnIndices[i], row - 1);
                    }

                    dependenciesResolved = newDependenciesResolved;
                }

                // dependencies have been resolved - we can calculate the value now;
                for (int i = 0; i < numSideColumns; i++) {
                    minPreviousEnergy =
                      min(minPreviousEnergy,
                        ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, sideColumnIndices[i], row - 1));
                }

                energy += minPreviousEnergy;
            }
            ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, col, row)       = energy;
            ENERGY_IMG_VAL(hasFinishedCalculationTable, originalWidth, col, row) = true;
        }
    }
} // calculateVerticalCumulativeEnergyTableKernel

__global__
void initializeHasFinishedCalculationTableKernel(
    bool * hasFinishedCalculationTable,
    int    hasFinishedCalculationTableSize
)
{
    int index  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < hasFinishedCalculationTableSize; i += stride) hasFinishedCalculationTable[i] = false;
}

int * getVerticalSeamColumnIndices(
    float * workingenergyMapArray,
    int originalWidth, int originalHeight,
    int currentWidth, int currentHeight)
{
    float * cumulativeEnergyTable;

    // TODO: Optimize to only use currentWidth and currentHeight
    hipMallocManaged(&cumulativeEnergyTable, originalWidth * originalHeight * sizeof(float));


    bool * hasFinishedCalculationTable;
    // TODO: Optimize to only use currentWidth and currentHeight
    hipMallocManaged(&hasFinishedCalculationTable, originalWidth * originalHeight * sizeof(bool));


    int blockSize = BLOCK_SIZE;
    int numBlocks = 1; // Don't split across multiple blocks to avoid memory access overhead

    initializeHasFinishedCalculationTableKernel << < numBlocks, blockSize >> > (
        hasFinishedCalculationTable,
        currentWidth * currentHeight
        );

    hipDeviceSynchronize();


    calculateVerticalCumulativeEnergyTableKernel << < numBlocks, blockSize >> > (
        workingenergyMapArray,
        hasFinishedCalculationTable,
        cumulativeEnergyTable,
        originalWidth,
        currentWidth, currentHeight
        );

    hipDeviceSynchronize();

    hipFree(hasFinishedCalculationTable);

    int * result;

    hipMallocManaged(&result, currentHeight * sizeof(int));


    int mRow = currentHeight - 1;
    int mCol = 0;
    for (int i = 0; i < currentWidth; i++) {
        if (ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, i, mRow)
          < ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, mCol, mRow))
        {
            mCol = i;
        }
    }
    result[mRow] = mCol;
    while (mRow > 0) {
        mRow--;

        int lCol = mCol - 1;
        int rCol = mCol + 1;
        if (lCol > 0 &&
          (ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, lCol, mRow)
          < ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, mCol, mRow)))
        {
            mCol = lCol;
        }
        if (rCol < originalWidth &&
          (ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, rCol, mRow)
          < ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, mCol, mRow)))
        {
            mCol = rCol;
        }
        result[mRow] = mCol;
    }

    hipFree(cumulativeEnergyTable);

    return result;
} // getVerticalSeamColumnIndices

__global__
void removeVerticalSeamKernel(
    float * workingArray,
    float * workingenergyMapArray,
    int * seamColumnIndices,
    int originalWidth,
    int currentWidth, int currentHeight
)
{
    int index  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int row = index; row < currentHeight; row += stride) {
        // remove seam for this row
        int targetColumn = seamColumnIndices[row];
        for (int i = targetColumn; i < currentWidth - 1; i++) {
            // shift over image pixel
            for (int color = 0; color < NUM_COLORS; color++) {
                IMG_VAL(workingArray, originalWidth, i, row, color) =
                  IMG_VAL(workingArray, originalWidth, i + 1, row, color);
            }

            // shift over energy map pixel
            ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, i, row) =
              ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, i + 1, row);

            // energy map is now invalid - it needs to be recalculated along the seam
            // do that outside the kernel to avoid race conditions
        }
    }
}

void removeVerticalSeam(
    float * workingArray,
    float * workingenergyMapArray,
    int * seamColumnIndices,
    int originalWidth, int originalHeight,
    int currentWidth, int currentHeight)
{
    int blockSize = BLOCK_SIZE;
    int numBlocks = 1; // Don't split across multiple blocks to avoid memory access overhead

    removeVerticalSeamKernel << < numBlocks, blockSize >> > (
        workingArray,
        workingenergyMapArray,
        seamColumnIndices,
        originalWidth,
        currentWidth, currentHeight
        );

    hipDeviceSynchronize();

    // energy map is now invalid - it needs to be recalculated along the seam
    // recalculate energy map values along seam
    currentWidth -= 1;
    for (int row = 0; row < currentHeight; row++) {
        int removedColumn = seamColumnIndices[row];
        if (removedColumn > 0) {
            ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, removedColumn - 1, row) = getPixelEnergy(
                workingArray, removedColumn - 1, row, originalWidth, originalHeight, currentWidth, currentHeight);
        }
        if (removedColumn < currentWidth - 1) {
            ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, removedColumn, row) = getPixelEnergy(
                workingArray, removedColumn, row, originalWidth, originalHeight, currentWidth, currentHeight);
        }
    }
}

void identifyAndRemoveVerticalSeam(
    float * workingArray,
    float * workingenergyMapArray,
    int originalWidth, int originalHeight,
    int currentWidth, int currentHeight)
{
    int * verticalSeamColumnIndices = getVerticalSeamColumnIndices(workingenergyMapArray,
        originalWidth, originalHeight, currentWidth, currentHeight);

    removeVerticalSeam(workingArray, workingenergyMapArray, verticalSeamColumnIndices,
      originalWidth, originalHeight,
      currentWidth, currentHeight);

    hipFree(verticalSeamColumnIndices);
}

__global__
void calculateHorizontalCumulativeEnergyTableKernel(
    float * workingenergyMapArray,
    bool * hasFinishedCalculationTable,
    float * cumulativeEnergyTable,
    int originalWidth,
    int currentWidth, int currentHeight
)
{
    int index  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int col = 0; col < currentWidth; col++) {
        // iterate over every row this kernel is responsible for
        for (int row = index; row < currentHeight; row += stride) {
            // calculate the minimum cumulative energy for this index;
            float energy = ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, col, row);
            if (col > 0) {
                float minPreviousEnergy     = ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, col - 1, row); // guaranteed to be done
                int numDependencyRows       = 0;
                int dependencyRowIndices[2] = { 0, 0 }; // placeholders
                if (row > 0) {
                    dependencyRowIndices[numDependencyRows] = row - 1;
                    numDependencyRows++;
                }
                if (row < currentHeight - 1) {
                    dependencyRowIndices[numDependencyRows] = row + 1;
                    numDependencyRows++;
                }
                bool dependenciesResolved = false;
                // keep checking to see if dependencies (adjacent rows) have been resolved
                while (!dependenciesResolved) {
                    bool newDependenciesResolved = true;
                    for (int i = 0; i < numDependencyRows; i++) {
                        newDependenciesResolved = newDependenciesResolved &&
                          ENERGY_IMG_VAL(hasFinishedCalculationTable, originalWidth, col - 1, dependencyRowIndices[i]);
                    }

                    dependenciesResolved = newDependenciesResolved;
                }

                // dependencies have been resolved - we can calculate the value now;
                for (int i = 0; i < numDependencyRows; i++) {
                    minPreviousEnergy =
                      min(minPreviousEnergy,
                        ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, col - 1, dependencyRowIndices[i]));
                }

                energy += minPreviousEnergy;
            }
            ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, col, row)       = energy;
            ENERGY_IMG_VAL(hasFinishedCalculationTable, originalWidth, col, row) = true;
        }
    }
} // calculateVerticalCumulativeEnergyTableKernel

int * getHorizontalSeamColumnIndices(
    float * workingenergyMapArray,
    int originalWidth, int originalHeight,
    int currentWidth, int currentHeight)
{
    float * cumulativeEnergyTable;

    // TODO: Optimize to only use currentWidth and currentHeight
    hipMallocManaged(&cumulativeEnergyTable, originalWidth * originalHeight * sizeof(float));


    bool * hasFinishedCalculationTable;
    // TODO: Optimize to only use currentWidth and currentHeight
    hipMallocManaged(&hasFinishedCalculationTable, originalWidth * originalHeight * sizeof(bool));


    int blockSize = BLOCK_SIZE;
    int numBlocks = 1; // Don't split across multiple blocks to avoid memory access overhead

    initializeHasFinishedCalculationTableKernel << < numBlocks, blockSize >> > (
        hasFinishedCalculationTable,
        currentWidth * currentHeight
        );

    hipDeviceSynchronize();


    calculateHorizontalCumulativeEnergyTableKernel << < numBlocks, blockSize >> > (
        workingenergyMapArray,
        hasFinishedCalculationTable,
        cumulativeEnergyTable,
        originalWidth,
        currentWidth, currentHeight
        );

    hipDeviceSynchronize();


    hipFree(hasFinishedCalculationTable);

    int * result;

    hipMallocManaged(&result, currentWidth * sizeof(int));

    int mRow = 0;
    int mCol = currentWidth - 1;
    for (int i = 0; i < currentHeight; i++) {
        if (ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, mCol, i)
          < ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, mCol, mRow))
        {
            mRow = i;
        }
    }
    result[mCol] = mRow;
    while (mCol > 0) {
        mCol--;

        int dRow = mRow - 1;
        int uRow = mRow + 1;
        if (dRow > 0 &&
          (ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, mCol, dRow)
          < ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, mCol, mRow)))
        {
            mRow = dRow;
        }
        if (uRow < originalHeight &&
          (ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, mCol, uRow)
          < ENERGY_IMG_VAL(cumulativeEnergyTable, originalWidth, mCol, mRow)))
        {
            mRow = uRow;
        }
        result[mCol] = mRow;
    }

    hipFree(cumulativeEnergyTable);

    return result;
} // getVerticalSeamColumnIndices

__global__
void removeHorizontalSeamKernel(
    float * workingArray,
    float * workingenergyMapArray,
    int * seamRowIndices,
    int originalWidth,
    int currentWidth, int currentHeight
)
{
    int index  = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int col = index; col < currentWidth; col += stride) {
        // remove seam for this row
        int targetRow = seamRowIndices[col];

        for (int i = targetRow; i < currentHeight - 1; i++) {
            // shift over image pixel
            for (int color = 0; color < NUM_COLORS; color++) {
                IMG_VAL(workingArray, originalWidth, col, i, color) =
                  IMG_VAL(workingArray, originalWidth, col, i + 1, color);
            }

            // shift over energy map pixel
            ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, col, i) =
              ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, col, i + 1);

            // energy map is now invalid - it needs to be recalculated along the seam
            // do that outside the kernel to avoid race conditions
        }
    }
}

void removeHorizontalSeam(
    float * workingArray,
    float * workingenergyMapArray,
    int * seamRowIndices,
    int originalWidth, int originalHeight,
    int currentWidth, int currentHeight)
{
    int blockSize = BLOCK_SIZE;
    int numBlocks = 1; // Don't split across multiple blocks to avoid memory access overhead

    removeHorizontalSeamKernel << < numBlocks, blockSize >> > (
        workingArray,
        workingenergyMapArray,
        seamRowIndices,
        originalWidth,
        currentWidth, currentHeight
        );

    hipDeviceSynchronize();

    // energy map is now invalid - it needs to be recalculated along the seam
    // recalculate energy map values along seam
    currentHeight -= 1;
    for (int col = 0; col < currentWidth; col++) {
        int removedRow = seamRowIndices[col];
        if (removedRow > 0) {
            ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, col, removedRow - 1) = getPixelEnergy(
                workingArray, col, removedRow - 1, originalWidth, originalHeight, currentWidth, currentHeight);
        }
        if (removedRow < currentHeight - 1) {
            ENERGY_IMG_VAL(workingenergyMapArray, originalWidth, col, removedRow) = getPixelEnergy(
                workingArray, col, removedRow, originalWidth, originalHeight, currentWidth, currentHeight);
        }
    }
}

void identifyAndRemoveHorizontalSeam(
    float * workingArray,
    float * workingenergyMapArray,
    int originalWidth, int originalHeight,
    int currentWidth, int currentHeight)
{
    int * seamRowIndices = getHorizontalSeamColumnIndices(workingenergyMapArray,
        originalWidth, originalHeight, currentWidth, currentHeight);

    removeHorizontalSeam(workingArray, workingenergyMapArray, seamRowIndices,
      originalWidth, originalHeight,
      currentWidth, currentHeight);

    hipFree(seamRowIndices);
}
