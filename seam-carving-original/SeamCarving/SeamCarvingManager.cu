#include <bits/stdc++.h>
#include <iostream>
#include <CImg.h>

#include "SeamCarvingShrinker.h"

using namespace cimg_library;


CImg<float> seamCarvingShrink(CImg<float> inputImage, int newWidth, int newHeight)
{
    int inputWidth  = inputImage.width();
    int inputHeight = inputImage.height();

    float * workingArray = getWorkingArray(inputImage);
    float * workingenergyMapArray = getEnergyMap(workingArray, inputWidth, inputHeight, inputWidth, inputHeight);

    float totalWidthShrinks  = inputWidth - newWidth;
    float totalHeightShrinks = inputHeight - newHeight;

    int currentWidth  = inputWidth;
    int currentHeight = inputHeight;

    while (currentWidth > newWidth || currentHeight > newHeight) {
        if ((currentWidth - newWidth) / totalWidthShrinks >
          (currentHeight - newHeight) / totalHeightShrinks)
        {
            identifyAndRemoveVerticalSeam(
                workingArray,
                workingenergyMapArray,
                inputWidth, inputHeight,
                currentWidth, currentHeight);
            currentWidth -= 1;
        } else {
            identifyAndRemoveHorizontalSeam(
                workingArray,
                workingenergyMapArray,
                inputWidth, inputHeight,
                currentWidth, currentHeight);
            currentHeight -= 1;
        }
    }

    CImg<float> outputImage =
      getOutputImage(workingArray, inputWidth, inputHeight, newWidth, newHeight);

    // CImg<float> outputEnergyMapImage =
    //   getOutputEnergyMapImage(workingenergyMapArray, inputWidth, newWidth, newHeight);


    hipFree(workingArray);
    hipFree(workingenergyMapArray);

    // CImgDisplay main_disp(outputEnergyMapImage);
    // std::cin.ignore();


    return outputImage;
} // seamCarvingShrink
